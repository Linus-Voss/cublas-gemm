/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <cblas.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define LIM_CHECK_N 4096
#define LIM_PRINT_N 32

// fraction error   1.0 is 100% 
#define TOLERR 0.0001

#include "tools.h"

using namespace std;

int main(int argc, char **argv) {
  hipblasStatus_t status;
  if(argc != 5){
      fprintf(stderr, "run as ./prog dev nt n comptype\n\n");
      printArgsInfo();
      return EXIT_FAILURE;
  }
  float gputime_ms;
  int dev = atoi(argv[1]);
  int nt = atoi(argv[2]);
  int N = atoi(argv[3]);
  int comptype = atoi(argv[4]);
  // host pointers
  ATYPE *h_A;
  float *cblasA;
  BTYPE *h_B;
  float *cblasB;
  CTYPE *h_C;
  float *cblasC;
  // device pointers
  ATYPE *d_A = 0;
  BTYPE *d_B = 0;
  CTYPE *d_C = 0;
  // constants
  CTYPE alpha = 1.0f;
  CTYPE beta = 0.0f;
  // number of elements
  unsigned long nelem = N * N;
  double t1, t2;
  double TFLOP = 2.0*N*N*N * 1E-12;
  int bitsA = sizeof(ATYPE)*8;
  int bitsB = sizeof(BTYPE)*8;
  int bitsC = sizeof(CTYPE)*8;

  hipDataType dtypeA = dataTypes[hmap(bitsA)];
  hipDataType dtypeB = dataTypes[hmap(bitsB)];
  hipDataType dtypeC = dataTypes[hmap(bitsC)];
  const char* dtypeAStr = dataTypesStr[hmap(bitsA)];
  const char* dtypeBStr = dataTypesStr[hmap(bitsB)];
  const char* dtypeCStr = dataTypesStr[hmap(bitsC)];

  hipSetDevice(dev);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipblasHandle_t handle;
  omp_set_num_threads(nt);
  printf("MATMUL A x B = C (%i x %i)\nA FP%i (%s)\nB FP%i (%s)\nC FP%i (%s)\n\n", 
          N, N,  
          bitsA, dtypeAStr,
          bitsB, dtypeBStr,
          bitsC, dtypeCStr);



  /* 1) Initialize CUBLAS */
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }


  /* 2) Set math mode */
  printf("Compute Type.....................%s\n", cublasComputeTypesStr[comptype]);
  //status = hipblasSetMathMode(handle, cublasMathModes[0]);
  if (status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "!!!! CUBLAS MATH MODE ERROR\n");
    return EXIT_FAILURE;
  }


  /* 3) Allocate and fill host memory for the matrices */
  printf("Host mallocs A B C............."); fflush(stdout);
  t1 = omp_get_wtime();
  //h_A = (ATYPE*)(malloc(nelem * sizeof(h_A[0])));
  //h_B = (BTYPE*)(malloc(nelem * sizeof(h_B[0])));
  //h_C = (CTYPE*)(malloc(nelem * sizeof(h_C[0])));

  #ifdef PINNED
      hipHostMalloc((void**)&h_A, nelem*sizeof(h_A[0]));
      hipHostMalloc((void**)&h_B, nelem*sizeof(h_B[0]));
      hipHostMalloc((void**)&h_C, nelem*sizeof(h_C[0]));
  #else
      h_A = (ATYPE*)(malloc(nelem * sizeof(h_A[0])));
      h_B = (BTYPE*)(malloc(nelem * sizeof(h_B[0])));
      h_C = (CTYPE*)(malloc(nelem * sizeof(h_C[0])));
  #endif

  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);
  printf("Filling matrices in Host......."); fflush(stdout);
  t1 = omp_get_wtime();
  fillMatrixRand<ATYPE>(h_A, nelem);
  fillMatrixRand<BTYPE>(h_B, nelem);
  fillMatrixRand<CTYPE>(h_C, nelem);
  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);
  print_matrix<ATYPE>(h_A, N, N, "MAT A");
  print_matrix<BTYPE>(h_B, N, N, "MAT B");


  /* 4) Allocate device memory for the matrices */
  printf("Device mallocs A B C..........."); fflush(stdout);
  t1 = omp_get_wtime();
  if (hipMalloc(reinterpret_cast<void **>(&d_A), nelem * sizeof(d_A[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
  }

  if (hipMalloc(reinterpret_cast<void **>(&d_B), nelem * sizeof(d_B[0])) != hipSuccess) {
    fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
    return EXIT_FAILURE;
  }

  if (hipMalloc(reinterpret_cast<void **>(&d_C), nelem * sizeof(d_C[0])) != hipSuccess) {
    fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);



  /* 5) Initialize the device matrices with the host matrices */
    printf("\n");
  #ifdef PINNED
    printf("[PINNED MEMORY]\n");
  #endif
  printf("Device -> Host memcpy A........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_A[0]), h_A, 1, d_A, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write A)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_A[0])/(1e9 * (t2-t1))); fflush(stdout);

  printf("Device -> Host memcpy B........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_B[0]), h_B, 1, d_B, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write B)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_B[0])/(1e9 * (t2-t1))); fflush(stdout);

  printf("Device -> Host memcpy C........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_C[0]), h_C, 1, d_C, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write C)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n\n", t2-t1, nelem*sizeof(h_C[0])/(1e9 * (t2-t1))); fflush(stdout);







  /* 6) GEMM -> GPU CUBLAS */
  printf("[CUBLAS] GPU GEMM.............."); fflush(stdout);
  gpuErrchk(hipEventRecord(start));
  status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                                    d_A, dtypeA, N,
                                    d_B, dtypeB, N,
                          &beta,    d_C, dtypeC, N, cublasComputeTypes[comptype],  CUBLAS_GEMM_DEFAULT_TENSOR_OP);
  if(status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "!!!! kernel execution error.\n");
    return EXIT_FAILURE;
  }
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipEventRecord(stop));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&gputime_ms, start, stop));
  double gpuTFLOPS = TFLOP/(gputime_ms/1000.0);
  printf("done: %f secs   [%f TFLOPS]\n", gputime_ms/1000.0, gpuTFLOPS); fflush(stdout);





  /* 7) GEMM -> CPU BASIC */
  //printf("[CBLAS] (float) Host mallocs A B C............."); fflush(stdout);
  t1 = omp_get_wtime();
  cblasA = (float*)(malloc(nelem * sizeof(cblasA[0])));
  cblasB = (float*)(malloc(nelem * sizeof(cblasB[0])));
  cblasC = (float*)(malloc(nelem * sizeof(cblasC[0])));
  t2 = omp_get_wtime();
  //printf("done: %f secs\n", t2-t1); fflush(stdout);
  //printf("[CBLAS] (float) Filling matrices in Host......."); fflush(stdout);
  t1 = omp_get_wtime();
  copyMatrix<float, ATYPE>(cblasA, h_A, N);
  copyMatrix<float, BTYPE>(cblasB, h_B, N);
  t2 = omp_get_wtime();
  //printf("done: %f secs\n", t2-t1); fflush(stdout);
  printf("[CBLAS] CPU GEMM..............."); fflush(stdout);
  t1 = omp_get_wtime();
  //cpuGemm(N, alpha, h_A, h_B, beta, h_C);
  cblas_sgemm(CblasColMajor,CblasNoTrans,CblasNoTrans,N,N,N,alpha,cblasA,N,cblasB,N,beta,cblasC,N);
  t2 = omp_get_wtime();
  double cpuTFLOPS = TFLOP/(t2-t1);
  printf("done: %f secs   [%f TFLOPS]\n\n", t2-t1, cpuTFLOPS); fflush(stdout);
  print_matrix<float>(cblasC, N, N, "RESULT MAT C (CPU)");





  /* 8) Read the result back */
  printf("Device -> Host memcpy C........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasGetVector(nelem, sizeof(h_C[0]), d_C, 1, h_C, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (read C)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_C[0])/(1e9*(t2-t1))); fflush(stdout);
  print_matrix<CTYPE>(h_C, N, N, "RESULT MAT C (GPU)");





  /* 9) Check result against reference */
  printf("Verify result.................."); fflush(stdout);
  t1 = omp_get_wtime();
  double maxError = computeMaxError(cblasC, h_C, N); 
  t2 = omp_get_wtime();
  printf("done: %f secs (maxError = %f%%, TOL = %f%%)\n%s\n\n", t2-t1,
          maxError*100.0, TOLERR*100.0, 
          maxError <= TOLERR ? (const char*)"pass" : (const char*) "failed"); fflush(stdout);






  /* 10) Memory clean up */
  #ifdef PINNED
      hipHostFree(h_A);
      hipHostFree(h_B);
      hipHostFree(h_C);
  #else
      free(h_A);
      free(h_B);
      free(h_C);
  #endif

  if (hipFree(d_A) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (A)\n");
    return EXIT_FAILURE;
  }
  if (hipFree(d_B) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (B)\n");
    return EXIT_FAILURE;
  }
  if (hipFree(d_C) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
  }

  /* 11) Shutdown */
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! shutdown error (A)\n");
    return EXIT_FAILURE;
  }
}
